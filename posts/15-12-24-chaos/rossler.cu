
#include <hip/hip_runtime.h>
#include <stdio.h>
 
const unsigned int STEPS = 10000;
const unsigned int TRAJECTORIES = 20;

struct Point {
    double x, y, z;
};

struct Image {
    unsigned char r, g, b;
};

__global__ void rossler(Point* points, double dt, int steps) {
    int i = blockIdx.x;
    for (int j = 1; j < steps; ++j) {
        Point prev_point = points[i * steps + j - 1];
        points[i * STEPS + j].x = prev_point.x + dt*(- prev_point.y - prev_point.z);
        points[i * STEPS + j].y = prev_point.y + dt*(prev_point.x + 0.2 * prev_point.y);
        points[i * STEPS + j].z = prev_point.z + dt*(0.2 + prev_point.z * (prev_point.x - 5.7));
    }
}

int main() {

    Point* host_points = (Point*) malloc(TRAJECTORIES * STEPS * sizeof(Point));
    Point* device_points;

    // intialize host_points with random [-10,10] values
    for (int i = 0; i < TRAJECTORIES; ++i) {
        host_points[i * STEPS + 0].x = (rand() / (double)RAND_MAX) * 40 - 20;
        host_points[i * STEPS + 0].y = (rand() / (double)RAND_MAX) * 40 - 20;
        host_points[i * STEPS + 0].z = (rand() / (double)RAND_MAX) * 40 - 20;
    }

    hipMalloc(&device_points, TRAJECTORIES * STEPS * sizeof(Point));
    hipMemcpy(device_points, host_points, TRAJECTORIES * STEPS * sizeof(Point), hipMemcpyHostToDevice);

    printf("initialization done\n");

    // launch kernel
    dim3 block(TRAJECTORIES);
    rossler<<<block, 1>>>(device_points, 0.01, STEPS);
    hipMemcpy(host_points, device_points, TRAJECTORIES * STEPS * sizeof(Point), hipMemcpyDeviceToHost);

    printf("kernel done\n");

    // write to file
    FILE* file = fopen("rossler.txt", "w");
    for (int i = 0; i < TRAJECTORIES; ++i) {
        for (int j = 0; j < STEPS; ++j) {
            fprintf(file, "%f %f %f\n", host_points[i * STEPS + j].x, host_points[i * STEPS + j].y, host_points[i * STEPS + j].z);
        }
    }

    printf("file written\n");

    return 0;
}
